#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#ifdef __cplusplus
extern "C" {
#endif

#define THREAD_COUNT 10

__global__ void sum_array(float *arrayIn, float *arrayOut, int N) {
    int idx = threadIdx.x;
    int local_n = N / THREAD_COUNT;
    arrayOut[idx] = 0;
    for (int i = local_n * idx; i < local_n * (idx + 1); ++i) {
        arrayOut[idx] += arrayIn[i];
    }
}


void cudaMemAlloc(long size) {
//TODO check separate mem allocation performance gain
}

void cudaMemFree() {
//TODO check separate mem allocation performance gain
}

float cudaSum(float *array, int length) {

    float *arrayIn_d, *arrayOut_d;

    hipMalloc((void **) &arrayIn_d, length * sizeof(float));
    hipMalloc((void **) &arrayOut_d, THREAD_COUNT * sizeof(float));

    hipMemcpy(arrayIn_d, array, (length * sizeof(float)), hipMemcpyHostToDevice);

    sum_array << < 1, THREAD_COUNT >> > (arrayIn_d, arrayOut_d, length);

    float sum = 0.0f;

    float *arrayOut_h = (float *) malloc(sizeof(float) * THREAD_COUNT);
    hipMemcpy(arrayOut_h, arrayOut_d, (THREAD_COUNT * sizeof(float)), hipMemcpyDeviceToHost);

    for (int i = 0; i < THREAD_COUNT; ++i) {
        sum += arrayOut_h[i];
    }

    hipFree(arrayOut_d);
    hipFree(arrayIn_d);
    free(arrayOut_h);
    return sum;
}

#ifdef __cplusplus
}
#endif