#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>

#ifdef __cplusplus
extern "C" {
#endif

#define THREAD_COUNT 10

__global__ void sum_array(float *arrayIn, float *arrayOut, int N) {
    int idx = threadIdx.x;
    int local_n = N / THREAD_COUNT;
    arrayOut[idx] = 0;
    for (int i = local_n * idx; i < local_n * (idx + 1); ++i) {
        arrayOut[idx] += arrayIn[i];
    }
}

__global__ void max_array(float *arrayIn, float *arrayOut, int N) {
    int idx = threadIdx.x;
    int local_n = N / THREAD_COUNT;
    arrayOut[idx] = -FLT_MAX;
    for (int i = local_n * idx; i < local_n * (idx + 1); ++i) {
        if (arrayOut[idx] < arrayIn[i]) {
            arrayOut[idx] = arrayIn[i];
        }
    }
}


void cudaMemAlloc(long size) {
//TODO check separate mem allocation performance gain
}

void cudaMemFree() {
//TODO check separate mem allocation performance gain
}

float cudaSum(float *array, int length) {

    float *arrayIn_d, *arrayOut_d;

    hipMalloc((void **) &arrayIn_d, length * sizeof(float));
    hipMalloc((void **) &arrayOut_d, THREAD_COUNT * sizeof(float));

    hipMemcpy(arrayIn_d, array, (length * sizeof(float)), hipMemcpyHostToDevice);

    sum_array << < 1, THREAD_COUNT >> > (arrayIn_d, arrayOut_d, length);

    float sum = 0.0f;

    float *arrayOut_h = (float *) malloc(sizeof(float) * THREAD_COUNT);
    hipMemcpy(arrayOut_h, arrayOut_d, (THREAD_COUNT * sizeof(float)), hipMemcpyDeviceToHost);

    for (int i = 0; i < THREAD_COUNT; ++i) {
        sum += arrayOut_h[i];
    }

    hipFree(arrayOut_d);
    hipFree(arrayIn_d);
    free(arrayOut_h);
    return sum;
}


float cudaMax(float *array, int length) {


    float *arrayIn_d, *arrayOut_d;

    hipMalloc((void **) &arrayIn_d, length * sizeof(float));
    hipMalloc((void **) &arrayOut_d, THREAD_COUNT * sizeof(float));

    hipMemcpy(arrayIn_d, array, (length * sizeof(float)), hipMemcpyHostToDevice);

    max_array << < 1, THREAD_COUNT >> > (arrayIn_d, arrayOut_d, length);

    float max = -FLT_MAX;

    float *arrayOut_h = (float *) malloc(sizeof(float) * THREAD_COUNT);
    hipMemcpy(arrayOut_h, arrayOut_d, (THREAD_COUNT * sizeof(float)), hipMemcpyDeviceToHost);

    for (int i = 0; i < THREAD_COUNT; ++i) {
        if (max < arrayOut_h[i]) {
            max = arrayOut_h[i];
        }
    }

    hipFree(arrayOut_d);
    hipFree(arrayIn_d);
    free(arrayOut_h);
    return max;
}

#ifdef __cplusplus
}
#endif